/*
 * Copyright (c) 2012, Kristopher Wuollett
 * All rights reserved.
 *
 * This file is part of kriswuollett/compute.
 *
 * kriswuollett/compute is free software: you can redistribute it and/or modify
 * it under the terms of the BSD 3-Clause License as written in the COPYING
 * file.
 */
#ifndef _CUDA_ADD_GPU_CU_
#define _CUDA_ADD_GPU_CU_

#include <hip/hip_runtime.h>

__global__
void add_gpu_kernel(float * a, float * b, float * c, int len)
{ 
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < len) c[i] = a[i] + b[i];
}

extern "C"
void add_gpu(dim3 &dimGrid, dim3 &dimBlock,
             float * a, float * b, float * c, int len);

void add_gpu(dim3 &dimGrid, dim3 &dimBlock,
             float * a, float * b, float * c, int len)
{
    add_gpu_kernel<<<dimGrid, dimBlock>>>(a, b, c, len);
}


#endif // define _CUDA_ADD_GPU_CU_
